#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__
void rgba_to_greyscale(unsigned char* dev_R, unsigned char* dev_G, unsigned char* dev_B, unsigned char* dev_OUT)
{
	int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

	dev_OUT[globalIdx] = .299f * (float)dev_R[globalIdx] + .587f * (float)dev_G[globalIdx] + .114f * (float)dev_B[globalIdx];
}

void rgba_to_greyscale_cpp(unsigned char* R, unsigned char* G, unsigned char* B, unsigned char* OUT,
	int numRows, int numCols) {

	unsigned char* dev_R;
	unsigned char* dev_G;
	unsigned char* dev_B;
	unsigned char* dev_OUT;

	int imageSize = numRows * numCols * sizeof(unsigned char);
	
	hipMalloc((void**)&dev_R, imageSize);
	hipMalloc((void**)&dev_G, imageSize);
	hipMalloc((void**)&dev_B, imageSize);
	hipMalloc((void**)&dev_OUT, imageSize);

	hipMemcpy(dev_R, R, imageSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_G, G, imageSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, imageSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_OUT, OUT, imageSize, hipMemcpyHostToDevice);

	int gridSize = (numRows * numCols / 1024) + 1;
	int blockSize = numRows * numCols < 1024 ? numRows * numCols : 1024;

	rgba_to_greyscale << <gridSize, blockSize >> > (dev_R, dev_G, dev_B, dev_OUT);

	hipMemcpy(OUT, dev_OUT, imageSize, hipMemcpyDeviceToHost);

	hipFree(dev_R);
	hipFree(dev_G);
	hipFree(dev_B);
	hipFree(dev_OUT);
}
